#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"

void startTiming(hipEvent_t *start, hipEvent_t *stop) {
    hipEventCreate(start);
    hipEventCreate(stop);
    hipEventRecord(*start);
}

float stopTiming(hipEvent_t *start, hipEvent_t *stop) {
    hipEventRecord(*stop);
    hipEventSynchronize(*stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, *start, *stop);
    hipEventDestroy(*start);
    hipEventDestroy(*stop);
    return milliseconds / 1000.0f;
}
